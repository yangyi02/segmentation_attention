#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {

template <typename Dtype>
__global__ void kernel_backward(const int num, const int dim,
            const Dtype* pred, const Dtype* target,
            Dtype* bottom_diff, Dtype* count_buffer,
            const Dtype* neg_weight,
            const Dtype* pos_weight, 
            const bool has_ignore_label, const int ignore_label) {
  CUDA_KERNEL_LOOP(index, num * dim) {
    int n = index / dim;
    int d = index % dim;
    int pos = n * dim + d;
    const int label_value = static_cast<int>(target[pos]);
    if (has_ignore_label && label_value == ignore_label) {
      bottom_diff[pos] = 0;
      count_buffer[pos] = 0;
    } else {
      bottom_diff[pos] = neg_weight[n] * pred[pos] * (1 - target[pos])
          - pos_weight[n] * target[pos] * (1 - pred[pos]);
      if (target[pos]) {
        count_buffer[pos] = pos_weight[n];
      } else {
        count_buffer[pos] = neg_weight[n];
      }
    }
  }
}

template <typename Dtype>
void SigmoidWeightedCrossEntropyLossLayer<Dtype>::Backward_gpu(
    const vector<Blob<Dtype>*>& top, const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[1]) {
    LOG(FATAL) << this->type()
               << " Layer cannot backpropagate to label inputs.";
  }
  if (bottom.size() > 2) {
    if (propagate_down[2]) {
      LOG(FATAL) << this->type()
                 << " Layer cannot backpropagate to loss weight.";
    }
  }
  if (propagate_down[0]) {
    const int nthreads = bottom[0]->count();
    const int count = bottom[0]->count();
    const int num   = bottom[0]->num();
    const int dim   = bottom[0]->count() / bottom[0]->num();
    const Dtype* pred = sigmoid_output_->gpu_data();
    const Dtype* target = bottom[1]->gpu_data();
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    const Dtype* pos_weight = positive_weights_.gpu_data();
    const Dtype* neg_weight = negative_weights_.gpu_data();
    Dtype batch_weight = 0;
    // Since this memory is never used for anything else,
    // we use to to avoid allocating new GPU memory.
    Dtype* count_buffer = sigmoid_output_->mutable_gpu_diff();
    
    const bool has_ignore_label = !ignore_label_.empty();
    int ignore_label = 0;
    if (has_ignore_label) {
      CHECK(ignore_label_.size() == 1)
          << "Current gpu implementation only takes one ignore label.";
      ignore_label = *ignore_label_.begin();
    }
    kernel_backward<Dtype><<<CAFFE_GET_BLOCKS(nthreads),
        CAFFE_CUDA_NUM_THREADS>>>(num, dim, pred, target,
                    bottom_diff, count_buffer,
                    neg_weight, pos_weight, 
                    has_ignore_label, ignore_label);
    const Dtype loss_weight = top[0]->cpu_diff()[0];
    if (normalize_) {
      caffe_gpu_asum(nthreads, count_buffer, &batch_weight);
      if (batch_weight == 0) {
        batch_weight = 1;
      }
      caffe_gpu_scal(count, loss_weight / batch_weight, bottom_diff);
    } else {
      caffe_gpu_scal(count, loss_weight / num, bottom_diff);
    }
  }
}

INSTANTIATE_LAYER_GPU_BACKWARD(SigmoidWeightedCrossEntropyLossLayer);


}  // namespace caffe
