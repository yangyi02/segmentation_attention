#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {

template <typename Dtype>
__global__ void UpsampleForward(const int nthreads, const Dtype* bottom_data,
    const int num, const int channels, const int height,
    const int width, const int upsampled_height, const int upsampled_width,
    const int upsample_ratio, Dtype* top_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    int w = index % width;
    int h = (index / width) % height;
    int c = (index / width / height) % channels;
    int n = index / width / height / channels;
    int hstart = h * upsample_ratio;
    int wstart = w * upsample_ratio;
    int hend = hstart + upsample_ratio;
    int wend = wstart + upsample_ratio;
    hstart = max(hstart, 0);
    wstart = max(wstart, 0);
    hend = min(hend, upsampled_height);
    wend = min(wend, upsampled_width);
    top_data += (n * channels + c) * upsampled_height * upsampled_width;
    for (int uh = hstart; uh < hend; ++uh) {
      for (int uw = wstart; uw < wend; ++uw) {
        top_data[uh * upsampled_width + uw] = bottom_data[index];
      }
    }
  }
}

template <typename Dtype>
void UpsampleLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  int count = bottom[0]->count();
  // NOLINT_NEXT_LINE(whitespace/operators)
  UpsampleForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, bottom_data, bottom[0]->num(), channels_,
      height_, width_, upsampled_height_, upsampled_width_, upsample_ratio_, top_data);
  CUDA_POST_KERNEL_CHECK;
}

template <typename Dtype>
__global__ void UpsampleBackward(const int nthreads, const Dtype* top_diff,
    const int num, const int channels, const int height,
    const int width, const int upsampled_height, const int upsampled_width,
    const int upsample_ratio, Dtype* bottom_diff) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    // find out the local index
    // find out the local offset
    int w = index % width;
    int h = (index / width) % height;
    int c = (index / width / height) % channels;
    int n = index / width / height / channels;
    int hstart = h * upsample_ratio;
    int wstart = w * upsample_ratio;
    int hend = hstart + upsample_ratio;
    int wend = wstart + upsample_ratio;
    hstart = max(hstart, 0);
    wstart = max(wstart, 0);
    hend = min(hend, upsampled_height);
    wend = min(wend, upsampled_width);
    top_diff += (n * channels + c) * upsampled_height * upsampled_width;
    for (int uh = hstart; uh < hend; ++uh) {
      for (int uw = wstart; uw < wend; ++uw) {
        bottom_diff[index] += top_diff[uh * upsampled_width + uw];
      }
    }
  }
}

template <typename Dtype>
void UpsampleLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  if (!propagate_down[0]) {
    return;
  }
  const Dtype* top_diff = top[0]->gpu_diff();
  Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
  const int count = bottom[0]->count();
  caffe_gpu_set(count, Dtype(0.), bottom_diff);
  UpsampleBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, top_diff, top[0]->num(), channels_,
      height_, width_, upsampled_height_, upsampled_width_, upsample_ratio_, bottom_diff);
  CUDA_POST_KERNEL_CHECK;
}


INSTANTIATE_LAYER_GPU_FUNCS(UpsampleLayer);


}  // namespace caffe
