#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {

template <typename Dtype>
__global__ void ResizeForward(const int nthreads, const Dtype* bottom_data,
    const int num, const int channels, const int height,
    const int width, const int resized_height, const int resized_width,
    const Dtype resize_ratio, Dtype* top_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    int rw = index % resized_width;
    int rh = (index / resized_width) % resized_height;
    int c = (index / resized_width / resized_height) % channels;
    int n = index / resized_width / resized_height / channels;
    int h = int(rh / resize_ratio);
    int w = int(rw / resize_ratio);
    bottom_data += (n * channels + c) * height * width;
    top_data[index] = bottom_data[h * width + w];
  }
}

template <typename Dtype>
void ResizeLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  int count = top[0]->count();
  // NOLINT_NEXT_LINE(whitespace/operators)
  ResizeForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, bottom_data, bottom[0]->num(), channels_,
      height_, width_, resized_height_, resized_width_, resize_ratio_, top_data);
  CUDA_POST_KERNEL_CHECK;
}

template <typename Dtype>
__global__ void ResizeBackward(const int nthreads, const Dtype* top_diff,
    const int num, const int channels, const int height,
    const int width, const int resized_height, const int resized_width,
    const Dtype resize_ratio, Dtype* bottom_diff) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    // find out the local index
    // find out the local offset
    int w = index % width;
    int h = (index / width) % height;
    int c = (index / width / height) % channels;
    int n = index / width / height / channels;
    int hstart = int(h * resize_ratio);
    int wstart = int(w * resize_ratio);
    int hend = int(hstart + resize_ratio);
    int wend = int(wstart + resize_ratio);
    hstart = max(hstart, 0);
    wstart = max(wstart, 0);
    hend = min(hend, resized_height);
    wend = min(wend, resized_width);
    top_diff += (n * channels + c) * resized_height * resized_width;
    for (int rh = hstart; rh < hend; ++rh) {
      for (int rw = wstart; rw < wend; ++rw) {
        bottom_diff[index] += top_diff[rh * resized_width + rw];
      }
    }
  }
}

template <typename Dtype>
void ResizeLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  if (!propagate_down[0]) {
    return;
  }
  const Dtype* top_diff = top[0]->gpu_diff();
  Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
  const int count = bottom[0]->count();
  caffe_gpu_set(count, Dtype(0.), bottom_diff);
  ResizeBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, top_diff, top[0]->num(), channels_,
      height_, width_, resized_height_, resized_width_, resize_ratio_, bottom_diff);
  CUDA_POST_KERNEL_CHECK;
}


INSTANTIATE_LAYER_GPU_FUNCS(ResizeLayer);


}  // namespace caffe
